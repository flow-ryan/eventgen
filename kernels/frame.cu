#include <hip/hip_runtime.h>
#include <iostream>
#include "frame.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"


int
Frame::load_from_file(std::string filename)
{
	this->filename = filename;
	// request the image as grayscale
	this->data = stbi_load(filename.c_str(), &this->x, &this->y, &this->n, 1);
	this->memsize = x * y * sizeof(unsigned char);
	if (!data) {
		// error happened, handle accordingly!
		std::cerr << "EE: File '" << filename << "' corrupt. Could not read data." << std::endl;
		return 1;
	}

	hipMalloc((void**)&dev_data, memsize);
	return 0;
}

void
Frame::toGPU()
{
	hipMemcpy(dev_data, data, memsize, hipMemcpyHostToDevice);
}

Frame::
~Frame() {
	if (dev_data) hipFree(dev_data);
	if (data) stbi_image_free(data);
}
